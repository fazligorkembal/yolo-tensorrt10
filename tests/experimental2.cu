#include <hip/hip_runtime.h>

#include <iostream>
#include <chrono>

// Basit dummy kernel - vektör toplama
__global__ void dummyKernel(float* d_a, float* d_b, float* d_c, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        // Basit bir işlem
        d_c[idx] = d_a[idx] + d_b[idx];
        
        // Biraz daha iş yükü ekleyelim
        for(int i = 0; i < 100; i++) {
            d_c[idx] = sin(d_c[idx]) + cos(d_a[idx]);
        }
    }
}

#define CHECK_CUDA(call) \
    do { \
        hipError_t error = call; \
        if (error != hipSuccess) { \
            std::cerr << "CUDA error at " << __FILE__ << ":" << __LINE__ \
                     << " - " << hipGetErrorString(error) << std::endl; \
            exit(1); \
        } \
    } while(0)

int main() {
    // RTX 4070 için optimize edilmiş parametreler
    const int N = 1024 * 1024;  // 1M eleman
    const int ITERATIONS = 6000;
    const int BLOCK_SIZE = 256;  // RTX 4070 için iyi bir değer
    const int GRID_SIZE = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;
    
    size_t bytes = N * sizeof(float);
    
    // Host pointers (pinned memory)
    float *h_a, *h_b, *h_c;
    
    // Pinned memory allocation
    CHECK_CUDA(hipHostMalloc(&h_a, bytes, hipHostMallocDefault));
    CHECK_CUDA(hipHostMalloc(&h_b, bytes, hipHostMallocDefault));
    CHECK_CUDA(hipHostMalloc(&h_c, bytes, hipHostMallocDefault));
    
    // Device pointers
    float *d_a, *d_b, *d_c;
    CHECK_CUDA(hipMalloc(&d_a, bytes));
    CHECK_CUDA(hipMalloc(&d_b, bytes));
    CHECK_CUDA(hipMalloc(&d_c, bytes));
    
    // Initialize host data
    for(int i = 0; i < N; i++) {
        h_a[i] = static_cast<float>(i);
        h_b[i] = static_cast<float>(i * 2);
    }
    
    // Stream oluştur
    hipStream_t stream;
    CHECK_CUDA(hipStreamCreate(&stream));
    
    // Async memory transfer (H2D)
    CHECK_CUDA(hipMemcpyAsync(d_a, h_a, bytes, hipMemcpyHostToDevice, stream));
    CHECK_CUDA(hipMemcpyAsync(d_b, h_b, bytes, hipMemcpyHostToDevice, stream));
    
    // CUDA Graph oluşturma
    hipGraph_t graph;
    hipGraphExec_t graphExec;
    
    // Graph capture başlat
    CHECK_CUDA(hipStreamBeginCapture(stream, hipStreamCaptureModeGlobal));
    
    // Kernel'i graph'a ekle
    dummyKernel<<<GRID_SIZE, BLOCK_SIZE, 0, stream>>>(d_a, d_b, d_c, N);
    
    // Graph capture bitir
    CHECK_CUDA(hipStreamEndCapture(stream, &graph));
    
    // Graph instance oluştur
    CHECK_CUDA(hipGraphInstantiate(&graphExec, graph, NULL, NULL, 0));
    
    // Warm-up
    for(int i = 0; i < 10; i++) {
        CHECK_CUDA(hipGraphLaunch(graphExec, stream));
    }
    CHECK_CUDA(hipStreamSynchronize(stream));
    
    // Performance ölçümü başlat
    auto start = std::chrono::high_resolution_clock::now();
    
    // 6000 iterasyon çalıştır
    for(int iter = 0; iter < ITERATIONS; iter++) {
        CHECK_CUDA(hipGraphLaunch(graphExec, stream));
    }
    
    // Stream'i bekle
    CHECK_CUDA(hipStreamSynchronize(stream));
    
    auto end = std::chrono::high_resolution_clock::now();
    
    // Async memory transfer (D2H)
    CHECK_CUDA(hipMemcpyAsync(h_c, d_c, bytes, hipMemcpyDeviceToHost, stream));
    CHECK_CUDA(hipStreamSynchronize(stream));
    
    // Performans sonuçları
    auto duration = std::chrono::duration_cast<std::chrono::microseconds>(end - start);
    double ms = duration.count() / 1000.0;
    
    std::cout << "=== PERFORMANS SONUÇLARI ===" << std::endl;
    std::cout << "Toplam süre: " << ms << " ms" << std::endl;
    std::cout << "İterasyon sayısı: " << ITERATIONS << std::endl;
    std::cout << "İterasyon başına süre: " << ms / ITERATIONS << " ms" << std::endl;
    std::cout << "Throughput: " << (ITERATIONS * 1000.0) / ms << " iter/s" << std::endl;
    
    // Veri boyutu ve bant genişliği
    double gb = (3.0 * N * sizeof(float) * ITERATIONS) / (1024.0 * 1024.0 * 1024.0);
    std::cout << "İşlenen veri: " << gb << " GB" << std::endl;
    std::cout << "Efektif bant genişliği: " << (gb * 1000.0) / ms << " GB/s" << std::endl;
    
    // Sonuçları kontrol et (opsiyonel)
    bool correct = true;
    for(int i = 0; i < 10; i++) {  // İlk 10 elemanı kontrol et
        float expected = h_a[i] + h_b[i];
        // sin/cos işlemleri nedeniyle tam eşitlik beklemiyoruz
        if(std::abs(h_c[i]) > 1e6) {  // Basit bir kontrol
            correct = false;
            break;
        }
    }
    std::cout << "Sonuç kontrolü: " << (correct ? "BAŞARILI" : "BAŞARISIZ") << std::endl;
    
    // Temizlik
    CHECK_CUDA(hipGraphDestroy(graph));
    CHECK_CUDA(hipGraphExecDestroy(graphExec));
    CHECK_CUDA(hipStreamDestroy(stream));
    
    CHECK_CUDA(hipFree(d_a));
    CHECK_CUDA(hipFree(d_b));
    CHECK_CUDA(hipFree(d_c));
    
    CHECK_CUDA(hipHostFree(h_a));
    CHECK_CUDA(hipHostFree(h_b));
    CHECK_CUDA(hipHostFree(h_c));
    
    std::cout << "\nProgram başarıyla tamamlandı!" << std::endl;
    
    return 0;
}

// Derleme komutu:
// nvcc -O3 -arch=sm_89 cuda_kernel.cu -o cuda_kernel
// RTX 4070 için sm_89 kullanıyoruz (Ada Lovelace)