#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>
#include <functional>
#include <cmath>
#include <mutex>
#include <thread>

// Görüntü boyutları
const int BATCH = 1;
const int CHANNELS = 3;
const int HEIGHT = 640;
const int WIDTH = 640;
const size_t IMAGE_SIZE = BATCH * CHANNELS * HEIGHT * WIDTH * sizeof(float);
const int TOTAL_IMAGES = 6000; // Tüm testlerde aynı toplam resim sayısı

// Ağırlaştırılmış dummy kernel
__global__ void dummyKernel(float *input, float *output, int size)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size)
    {
        float val = input[idx];
        // Daha fazla işlem ekleyelim
        for (int i = 0; i < 100; i++)
        {
            val = sinf(val) * cosf(val);
        }
        output[idx] = val;
    }
}

// Zaman ölçme fonksiyonu (iterasyonlu)
float measureTime(std::function<void()> func, int iterations)
{
    auto start = std::chrono::high_resolution_clock::now();
    for (int i = 0; i < iterations; i++)
    {
        func();
    }
    hipDeviceSynchronize();
    auto end = std::chrono::high_resolution_clock::now();
    return std::chrono::duration<float, std::milli>(end - start).count();
}

void runSynchronous()
{
    float *h_input = new float[IMAGE_SIZE / sizeof(float)];
    float *h_output = new float[IMAGE_SIZE / sizeof(float)];
    float *d_input, *d_output;

    hipMalloc(&d_input, IMAGE_SIZE);
    hipMalloc(&d_output, IMAGE_SIZE);

    // Veriyi hazırla
    for (size_t i = 0; i < IMAGE_SIZE / sizeof(float); i++)
    {
        h_input[i] = static_cast<float>(i);
    }

    int iterations = TOTAL_IMAGES; // 60,000 iterasyon
    auto time = measureTime([&]()
                            {
        hipMemcpy(d_input, h_input, IMAGE_SIZE, hipMemcpyHostToDevice);
        
        int blockSize = 256;
        int gridSize = (IMAGE_SIZE / sizeof(float) + blockSize - 1) / blockSize;
        dummyKernel<<<gridSize, blockSize>>>(d_input, d_output, IMAGE_SIZE / sizeof(float));
        
        hipMemcpy(h_output, d_output, IMAGE_SIZE, hipMemcpyDeviceToHost); }, iterations);

    float throughput = (iterations * 1000.0f) / time; // resim/saniye
    std::cout << "Senkronize - Toplam Süre: " << time << " ms, Throughput: " << throughput << " img/s\n";

    // Temizlik
    delete[] h_input;
    delete[] h_output;
    hipFree(d_input);
    hipFree(d_output);
}

void runAsynchronous()
{
    float *h_input, *h_output;
    hipHostAlloc(&h_input, IMAGE_SIZE, hipHostMallocDefault);
    hipHostAlloc(&h_output, IMAGE_SIZE, hipHostMallocDefault);

    float *d_input, *d_output;
    hipMalloc(&d_input, IMAGE_SIZE);
    hipMalloc(&d_output, IMAGE_SIZE);

    // Veriyi hazırla
    for (size_t i = 0; i < IMAGE_SIZE / sizeof(float); i++)
    {
        h_input[i] = static_cast<float>(i);
    }

    hipStream_t stream;
    hipStreamCreate(&stream);

    int iterations = TOTAL_IMAGES; // 60,000 iterasyon
    auto time = measureTime([&]()
                            {
        hipMemcpyAsync(d_input, h_input, IMAGE_SIZE, hipMemcpyHostToDevice, stream);
        
        int blockSize = 256;
        int gridSize = (IMAGE_SIZE / sizeof(float) + blockSize - 1) / blockSize;
        dummyKernel<<<gridSize, blockSize, 0, stream>>>(d_input, d_output, IMAGE_SIZE / sizeof(float));
        
        hipMemcpyAsync(h_output, d_output, IMAGE_SIZE, hipMemcpyDeviceToHost, stream); }, iterations);

    float throughput = (iterations * 1000.0f) / time;
    std::cout << "Asenkron - Toplam Süre: " << time << " ms, Throughput: " << throughput << " img/s\n";

    // Temizlik
    hipHostFree(h_input);
    hipHostFree(h_output);
    hipFree(d_input);
    hipFree(d_output);
    hipStreamDestroy(stream);
}

void runCUDAGraph()
{
    float *h_input, *h_output;
    hipHostAlloc(&h_input, IMAGE_SIZE, hipHostMallocDefault);
    hipHostAlloc(&h_output, IMAGE_SIZE, hipHostMallocDefault);

    float *d_input, *d_output;
    hipMalloc(&d_input, IMAGE_SIZE);
    hipMalloc(&d_output, IMAGE_SIZE);

    // Veriyi hazırla
    for (size_t i = 0; i < IMAGE_SIZE / sizeof(float); i++)
    {
        h_input[i] = static_cast<float>(i);
    }

    hipStream_t stream;
    hipStreamCreate(&stream);

    // CUDA Graph oluşturma
    hipGraph_t graph;
    hipGraphExec_t graphExec;

    hipStreamBeginCapture(stream, hipStreamCaptureModeGlobal);
    hipMemcpyAsync(d_input, h_input, IMAGE_SIZE, hipMemcpyHostToDevice, stream);

    int blockSize = 256;
    int gridSize = (IMAGE_SIZE / sizeof(float) + blockSize - 1) / blockSize;
    dummyKernel<<<gridSize, blockSize, 0, stream>>>(d_input, d_output, IMAGE_SIZE / sizeof(float));

    hipMemcpyAsync(h_output, d_output, IMAGE_SIZE, hipMemcpyDeviceToHost, stream);
    hipStreamEndCapture(stream, &graph);
    hipGraphInstantiate(&graphExec, graph, NULL, NULL, 0);

    int iterations = TOTAL_IMAGES; // 60,000 iterasyon
    auto time = measureTime([&]()
                            { hipGraphLaunch(graphExec, stream); }, iterations);

    float throughput = (iterations * 1000.0f) / time;
    std::cout << "CUDA Graph - Toplam Süre: " << time << " ms, Throughput: " << throughput << " img/s\n";

    // Temizlik
    hipHostFree(h_input);
    hipHostFree(h_output);
    hipFree(d_input);
    hipFree(d_output);
    hipStreamDestroy(stream);
    hipGraphExecDestroy(graphExec);
    hipGraphDestroy(graph);
}

void runMultiStream()
{
    const int IMAGES_PER_ITER = 3;
    const int ITERATIONS = TOTAL_IMAGES / IMAGES_PER_ITER; // 20,000 iterasyon

    float *h_input[IMAGES_PER_ITER], *h_output[IMAGES_PER_ITER];
    float *d_input[IMAGES_PER_ITER], *d_output[IMAGES_PER_ITER];
    hipStream_t streams[IMAGES_PER_ITER];

    // Bellek ayırma
    for (int i = 0; i < IMAGES_PER_ITER; i++)
    {
        hipHostAlloc(&h_input[i], IMAGE_SIZE, hipHostMallocDefault);
        hipHostAlloc(&h_output[i], IMAGE_SIZE, hipHostMallocDefault);
        hipMalloc(&d_input[i], IMAGE_SIZE);
        hipMalloc(&d_output[i], IMAGE_SIZE);
        hipStreamCreate(&streams[i]);

        // Veriyi hazırla
        for (size_t j = 0; j < IMAGE_SIZE / sizeof(float); j++)
        {
            h_input[i][j] = static_cast<float>(j);
        }
    }

    auto time = measureTime([&]()
                            {
        for (int i = 0; i < IMAGES_PER_ITER; i++) {
            hipMemcpyAsync(d_input[i], h_input[i], IMAGE_SIZE, hipMemcpyHostToDevice, streams[i]);
            
            int blockSize = 256;
            int gridSize = (IMAGE_SIZE / sizeof(float) + blockSize - 1) / blockSize;
            dummyKernel<<<gridSize, blockSize, 0, streams[i]>>>(d_input[i], d_output[i], IMAGE_SIZE / sizeof(float));
            
            hipMemcpyAsync(h_output[i], d_output[i], IMAGE_SIZE, hipMemcpyDeviceToHost, streams[i]);
        } }, ITERATIONS);

    float throughput = (TOTAL_IMAGES * 1000.0f) / time;
    std::cout << "Çoklu Stream - Toplam Süre: " << time << " ms, Throughput: " << throughput << " img/s\n";

    // Temizlik
    for (int i = 0; i < IMAGES_PER_ITER; i++)
    {
        hipHostFree(h_input[i]);
        hipHostFree(h_output[i]);
        hipFree(d_input[i]);
        hipFree(d_output[i]);
        hipStreamDestroy(streams[i]);
    }
}

void runOptimized()
{
    const int IMAGES_PER_ITER = 3;
    const int ITERATIONS = TOTAL_IMAGES / IMAGES_PER_ITER; // 20,000 iterasyon

    float *h_input[IMAGES_PER_ITER], *h_output[IMAGES_PER_ITER];
    float *d_input[IMAGES_PER_ITER], *d_output[IMAGES_PER_ITER];
    hipStream_t stream;
    hipGraph_t graph;
    hipGraphExec_t graphExec;

    hipStreamCreate(&stream);

    // Bellek ayırma
    for (int i = 0; i < IMAGES_PER_ITER; i++)
    {
        hipHostAlloc(&h_input[i], IMAGE_SIZE, hipHostMallocDefault);
        hipHostAlloc(&h_output[i], IMAGE_SIZE, hipHostMallocDefault);
        hipMalloc(&d_input[i], IMAGE_SIZE);
        hipMalloc(&d_output[i], IMAGE_SIZE);

        // Veriyi hazırla
        for (size_t j = 0; j < IMAGE_SIZE / sizeof(float); j++)
        {
            h_input[i][j] = static_cast<float>(j);
        }
    }

    // Tek graph oluştur
    hipStreamBeginCapture(stream, hipStreamCaptureModeGlobal);
    for (int i = 0; i < IMAGES_PER_ITER; i++)
    {
        hipMemcpyAsync(d_input[i], h_input[i], IMAGE_SIZE, hipMemcpyHostToDevice, stream);

        int blockSize = 256;
        int gridSize = (IMAGE_SIZE / sizeof(float) + blockSize - 1) / blockSize;
        dummyKernel<<<gridSize, blockSize, 0, stream>>>(d_input[i], d_output[i], IMAGE_SIZE / sizeof(float));

        hipMemcpyAsync(h_output[i], d_output[i], IMAGE_SIZE, hipMemcpyDeviceToHost, stream);
    }
    hipStreamEndCapture(stream, &graph);
    hipGraphInstantiate(&graphExec, graph, NULL, NULL, 0);

    auto time = measureTime([&]()
                            { hipGraphLaunch(graphExec, stream); }, ITERATIONS);

    float throughput = (TOTAL_IMAGES * 1000.0f) / time;
    std::cout << "Optimize Edilmiş - Toplam Süre: " << time << " ms, Throughput: " << throughput << " img/s\n";

    // Temizlik
    for (int i = 0; i < IMAGES_PER_ITER; i++)
    {
        hipHostFree(h_input[i]);
        hipHostFree(h_output[i]);
        hipFree(d_input[i]);
        hipFree(d_output[i]);
    }
    hipStreamDestroy(stream);
    hipGraphExecDestroy(graphExec);
    hipGraphDestroy(graph);
}

void runOptimizedPipeline() {
    const int NUM_STREAMS = 3;
    const int ITERATIONS = 6000 / NUM_STREAMS;
    
    float *h_input[NUM_STREAMS], *h_output[NUM_STREAMS];
    float *d_input[NUM_STREAMS], *d_output[NUM_STREAMS];
    hipStream_t streams[NUM_STREAMS];
    hipEvent_t startEvents[NUM_STREAMS], endEvents[NUM_STREAMS];
    
    // Bellek ayırma
    for (int i = 0; i < NUM_STREAMS; i++) {
        hipHostAlloc(&h_input[i], IMAGE_SIZE, hipHostMallocDefault);
        hipHostAlloc(&h_output[i], IMAGE_SIZE, hipHostMallocDefault);
        hipMalloc(&d_input[i], IMAGE_SIZE);
        hipMalloc(&d_output[i], IMAGE_SIZE);
        hipStreamCreate(&streams[i]);
        hipEventCreate(&startEvents[i]);
        hipEventCreate(&endEvents[i]);
        
        // Veriyi hazırla
        for (size_t j = 0; j < IMAGE_SIZE / sizeof(float); j++) {
            h_input[i][j] = static_cast<float>(j);
        }
    }

    // Zaman ölçümü
    hipEvent_t totalStart, totalEnd;
    hipEventCreate(&totalStart);
    hipEventCreate(&totalEnd);
    hipEventRecord(totalStart);
    
    // Pipeline başlat
    for (int i = 0; i < NUM_STREAMS; i++) {
        hipMemcpyAsync(d_input[i], h_input[i], IMAGE_SIZE, hipMemcpyHostToDevice, streams[i]);
        hipEventRecord(startEvents[i], streams[i]);
    }
    
    // Ana döngü
    for (int iter = 0; iter < ITERATIONS; iter++) {
        for (int i = 0; i < NUM_STREAMS; i++) {
            // Kernel çalıştır
            int blockSize = 256;
            int gridSize = (IMAGE_SIZE / sizeof(float) + blockSize - 1) / blockSize;
            dummyKernel<<<gridSize, blockSize, 0, streams[i]>>>(d_input[i], d_output[i], IMAGE_SIZE / sizeof(float));
            
            // D2H transferini başlat
            hipMemcpyAsync(h_output[i], d_output[i], IMAGE_SIZE, hipMemcpyDeviceToHost, streams[i]);
            hipEventRecord(endEvents[i], streams[i]);
            
            // Bir sonraki iterasyon için H2D transferini başlat
            if (iter < ITERATIONS - 1) {
                // D2H'nin bitmesini beklemeden H2D'yi başlat
                hipMemcpyAsync(d_input[i], h_input[i], IMAGE_SIZE, hipMemcpyHostToDevice, streams[i]);
                hipEventRecord(startEvents[i], streams[i]);
            }
        }
    }
    
    // Tüm stream'lerin bitmesini bekle
    for (int i = 0; i < NUM_STREAMS; i++) {
        hipStreamSynchronize(streams[i]);
    }
    
    hipEventRecord(totalEnd);
    hipEventSynchronize(totalEnd);
    
    float totalTime;
    hipEventElapsedTime(&totalTime, totalStart, totalEnd);
    
    float throughput = (6000 * 1000.0f) / totalTime;
    std::cout << "Optimize Pipeline - Toplam Süre: " << totalTime << " ms, Throughput: " << throughput << " img/s\n";
    
    // Temizlik
    for (int i = 0; i < NUM_STREAMS; i++) {
        hipHostFree(h_input[i]);
        hipHostFree(h_output[i]);
        hipFree(d_input[i]);
        hipFree(d_output[i]);
        hipStreamDestroy(streams[i]);
        hipEventDestroy(startEvents[i]);
        hipEventDestroy(endEvents[i]);
    }
    hipEventDestroy(totalStart);
    hipEventDestroy(totalEnd);
}

void runOptimizedGraph() {
    const int NUM_STREAMS = 3;
    const int ITERATIONS = 6000 / NUM_STREAMS;
    
    float *h_input[NUM_STREAMS], *h_output[NUM_STREAMS];
    float *d_input[NUM_STREAMS], *d_output[NUM_STREAMS];
    hipStream_t streams[NUM_STREAMS];
    hipGraph_t graphs[NUM_STREAMS];
    hipGraphExec_t graphExecs[NUM_STREAMS];
    
    // Bellek ayırma
    for (int i = 0; i < NUM_STREAMS; i++) {
        hipHostAlloc(&h_input[i], IMAGE_SIZE, hipHostMallocDefault);
        hipHostAlloc(&h_output[i], IMAGE_SIZE, hipHostMallocDefault);
        hipMalloc(&d_input[i], IMAGE_SIZE);
        hipMalloc(&d_output[i], IMAGE_SIZE);
        hipStreamCreate(&streams[i]);
        
        // Veriyi hazırla
        for (size_t j = 0; j < IMAGE_SIZE / sizeof(float); j++) {
            h_input[i][j] = static_cast<float>(j);
        }
        
        // Her stream için graph oluştur (sadece bir kez!)
        hipStreamBeginCapture(streams[i], hipStreamCaptureModeGlobal);
        hipMemcpyAsync(d_input[i], h_input[i], IMAGE_SIZE, hipMemcpyHostToDevice, streams[i]);
        
        int blockSize = 256;
        int gridSize = (IMAGE_SIZE / sizeof(float) + blockSize - 1) / blockSize;
        dummyKernel<<<gridSize, blockSize, 0, streams[i]>>>(d_input[i], d_output[i], IMAGE_SIZE / sizeof(float));
        
        hipMemcpyAsync(h_output[i], d_output[i], IMAGE_SIZE, hipMemcpyDeviceToHost, streams[i]);
        hipStreamEndCapture(streams[i], &graphs[i]);
        hipGraphInstantiate(&graphExecs[i], graphs[i], NULL, NULL, 0);
    }

    // Zaman ölçümü
    hipEvent_t totalStart, totalEnd;
    hipEventCreate(&totalStart);
    hipEventCreate(&totalEnd);
    hipEventRecord(totalStart);
    
    // Ana döngü
    for (int iter = 0; iter < ITERATIONS; iter++) {
        // Tüm stream'leri aynı anda başlat
        for (int i = 0; i < NUM_STREAMS; i++) {
            hipGraphLaunch(graphExecs[i], streams[i]);
        }
        
        // Bir sonraki iterasyon için veriyi hazırla
        if (iter < ITERATIONS - 1) {
            for (int i = 0; i < NUM_STREAMS; i++) {
                // Veriyi güncelle (CPU üzerinde)
                for (size_t j = 0; j < IMAGE_SIZE / sizeof(float); j++) {
                    h_input[i][j] = static_cast<float>(j + iter + 1);
                }
            }
        }
    }
    
    // Tüm stream'lerin bitmesini bekle
    for (int i = 0; i < NUM_STREAMS; i++) {
        hipStreamSynchronize(streams[i]);
    }
    
    hipEventRecord(totalEnd);
    hipEventSynchronize(totalEnd);
    
    float totalTime;
    hipEventElapsedTime(&totalTime, totalStart, totalEnd);
    
    float throughput = (6000 * 1000.0f) / totalTime;
    std::cout << "Optimize Graph - Toplam Süre: " << totalTime << " ms, Throughput: " << throughput << " img/s\n";
    
    // Temizlik
    for (int i = 0; i < NUM_STREAMS; i++) {
        hipHostFree(h_input[i]);
        hipHostFree(h_output[i]);
        hipFree(d_input[i]);
        hipFree(d_output[i]);
        hipStreamDestroy(streams[i]);
        hipGraphExecDestroy(graphExecs[i]);
        hipGraphDestroy(graphs[i]);
    }
    hipEventDestroy(totalStart);
    hipEventDestroy(totalEnd);
}

// Pinned bellek ile daha hızlı transfer
void optimizeMemoryTransfers() {
    // Unified Memory kullanımı alternatifi
    float *unified_input, *unified_output;
    hipMallocManaged(&unified_input, IMAGE_SIZE);
    hipMallocManaged(&unified_output, IMAGE_SIZE);
    
    // Veriyi hazırla
    for (size_t i = 0; i < IMAGE_SIZE / sizeof(float); i++) {
        unified_input[i] = static_cast<float>(i);
    }
    
    // Prefetch ile GPU'ya veriyi önceden getir
    int deviceId;
    hipGetDevice(&deviceId);
    hipMemPrefetchAsync(unified_input, IMAGE_SIZE, deviceId);
    hipMemPrefetchAsync(unified_output, IMAGE_SIZE, deviceId);
    
    // Kernel çalıştır
    int blockSize = 256;
    int gridSize = (IMAGE_SIZE / sizeof(float) + blockSize - 1) / blockSize;
    dummyKernel<<<gridSize, blockSize>>>(unified_input, unified_output, IMAGE_SIZE / sizeof(float));
    
    // Sonuçları CPU'ya geri getir
    hipMemPrefetchAsync(unified_output, IMAGE_SIZE, hipCpuDeviceId);
    hipDeviceSynchronize();
    
    // Temizlik
    hipFree(unified_input);
    hipFree(unified_output);
}

void runOptimizedPipelineFixed() {
    const int NUM_STREAMS = 3;
    const int ITERATIONS = 6000 / NUM_STREAMS;
    
    float *h_input[NUM_STREAMS], *h_output[NUM_STREAMS];
    float *d_input[NUM_STREAMS], *d_output[NUM_STREAMS];
    hipStream_t streams[NUM_STREAMS];
    
    // Bellek ayırma (event olmadan)
    for (int i = 0; i < NUM_STREAMS; i++) {
        hipHostAlloc(&h_input[i], IMAGE_SIZE, hipHostMallocDefault);
        hipHostAlloc(&h_output[i], IMAGE_SIZE, hipHostMallocDefault);
        hipMalloc(&d_input[i], IMAGE_SIZE);
        hipMalloc(&d_output[i], IMAGE_SIZE);
        hipStreamCreate(&streams[i]);
        
        // Veriyi hazırla
        for (size_t j = 0; j < IMAGE_SIZE / sizeof(float); j++) {
            h_input[i][j] = static_cast<float>(j);
        }
    }

    // Zaman ölçümü için sadece 2 event
    hipEvent_t totalStart, totalEnd;
    hipEventCreate(&totalStart);
    hipEventCreate(&totalEnd);
    hipEventRecord(totalStart);
    
    // İlk iterasyon için H2D transferlerini başlat
    for (int i = 0; i < NUM_STREAMS; i++) {
        hipMemcpyAsync(d_input[i], h_input[i], IMAGE_SIZE, hipMemcpyHostToDevice, streams[i]);
    }
    
    // Ana döngü - event kullanmadan
    for (int iter = 0; iter < ITERATIONS; iter++) {
        for (int i = 0; i < NUM_STREAMS; i++) {
            // Kernel çalıştır
            int blockSize = 256;
            int gridSize = (IMAGE_SIZE / sizeof(float) + blockSize - 1) / blockSize;
            dummyKernel<<<gridSize, blockSize, 0, streams[i]>>>(d_input[i], d_output[i], IMAGE_SIZE / sizeof(float));
            
            // D2H transferini başlat
            hipMemcpyAsync(h_output[i], d_output[i], IMAGE_SIZE, hipMemcpyDeviceToHost, streams[i]);
            
            // Bir sonraki iterasyon için H2D transferini başlat
            if (iter < ITERATIONS - 1) {
                // Event beklemeden doğrudan başlat
                hipMemcpyAsync(d_input[i], h_input[i], IMAGE_SIZE, hipMemcpyHostToDevice, streams[i]);
            }
        }
    }
    
    // Tüm stream'lerin bitmesini bekle
    for (int i = 0; i < NUM_STREAMS; i++) {
        hipStreamSynchronize(streams[i]);
    }
    
    hipEventRecord(totalEnd);
    hipEventSynchronize(totalEnd);
    
    float totalTime;
    hipEventElapsedTime(&totalTime, totalStart, totalEnd);
    
    float throughput = (6000 * 1000.0f) / totalTime;
    std::cout << "Optimize Pipeline (Fixed) - Toplam Süre: " << totalTime << " ms, Throughput: " << throughput << " img/s\n";
    
    // Temizlik
    for (int i = 0; i < NUM_STREAMS; i++) {
        hipHostFree(h_input[i]);
        hipHostFree(h_output[i]);
        hipFree(d_input[i]);
        hipFree(d_output[i]);
        hipStreamDestroy(streams[i]);
    }
    hipEventDestroy(totalStart);
    hipEventDestroy(totalEnd);
}





std::mutex mtx;

void processStream(int streamId, float* h_input, float* h_output, float* d_input, float* d_output, 
                  hipStream_t stream, int iterations, int* counter) {
    for (int iter = 0; iter < iterations; iter++) {
        // H2D transferi
        hipMemcpyAsync(d_input, h_input, IMAGE_SIZE, hipMemcpyHostToDevice, stream);
        
        // Kernel çalıştır
        int blockSize = 256;
        int gridSize = (IMAGE_SIZE / sizeof(float) + blockSize - 1) / blockSize;
        dummyKernel<<<gridSize, blockSize, 0, stream>>>(d_input, d_output, IMAGE_SIZE / sizeof(float));
        
        // D2H transferi
        hipMemcpyAsync(h_output, d_output, IMAGE_SIZE, hipMemcpyDeviceToHost, stream);
        
        // İlerlemeyi güncelle
        {
            std::lock_guard<std::mutex> lock(mtx);
            (*counter)++;
        }
    }
}

void runMultiThreadedPipeline() {
    const int NUM_STREAMS = 6; // Stream sayısını artırdık
    const int ITERATIONS = 6000 / NUM_STREAMS;
    
    float *h_input[NUM_STREAMS], *h_output[NUM_STREAMS];
    float *d_input[NUM_STREAMS], *d_output[NUM_STREAMS];
    hipStream_t streams[NUM_STREAMS];
    
    // Bellek ayırma
    for (int i = 0; i < NUM_STREAMS; i++) {
        hipHostAlloc(&h_input[i], IMAGE_SIZE, hipHostMallocDefault);
        hipHostAlloc(&h_output[i], IMAGE_SIZE, hipHostMallocDefault);
        hipMalloc(&d_input[i], IMAGE_SIZE);
        hipMalloc(&d_output[i], IMAGE_SIZE);
        hipStreamCreate(&streams[i]);
        
        // Veriyi hazırla
        for (size_t j = 0; j < IMAGE_SIZE / sizeof(float); j++) {
            h_input[i][j] = static_cast<float>(j);
        }
    }

    // Zaman ölçümü
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    
    // Thread'leri oluştur
    std::vector<std::thread> threads;
    int counter = 0;
    
    for (int i = 0; i < NUM_STREAMS; i++) {
        threads.emplace_back(processStream, i, h_input[i], h_output[i], 
                            d_input[i], d_output[i], streams[i], ITERATIONS, &counter);
    }
    
    // Thread'lerin bitmesini bekle
    for (auto& t : threads) {
        t.join();
    }
    
    // Tüm stream'lerin bitmesini bekle
    for (int i = 0; i < NUM_STREAMS; i++) {
        hipStreamSynchronize(streams[i]);
    }
    
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    
    float totalTime;
    hipEventElapsedTime(&totalTime, start, stop);
    
    float throughput = (6000 * 1000.0f) / totalTime;
    std::cout << "Multi-Threaded Pipeline - Toplam Süre: " << totalTime << " ms, Throughput: " << throughput << " img/s\n";
    std::cout << "Toplam işlenen resim: " << counter << "\n";
    
    // Temizlik
    for (int i = 0; i < NUM_STREAMS; i++) {
        hipHostFree(h_input[i]);
        hipHostFree(h_output[i]);
        hipFree(d_input[i]);
        hipFree(d_output[i]);
        hipStreamDestroy(streams[i]);
    }
    hipEventDestroy(start);
    hipEventDestroy(stop);
}

void runBatchProcessing() {
    const int BATCH_SIZE = 4; // Bir seferde 4 resim işle
    const int NUM_BATCHES = 6000 / BATCH_SIZE;
    
    float *h_input, *h_output;
    float *d_input, *d_output;
    hipStream_t stream;
    
    // Bellek ayırma (batch boyutunda)
    size_t batch_size = BATCH_SIZE * IMAGE_SIZE;
    hipHostAlloc(&h_input, batch_size, hipHostMallocDefault);
    hipHostAlloc(&h_output, batch_size, hipHostMallocDefault);
    hipMalloc(&d_input, batch_size);
    hipMalloc(&d_output, batch_size);
    hipStreamCreate(&stream);
    
    // Veriyi hazırla
    for (size_t i = 0; i < batch_size / sizeof(float); i++) {
        h_input[i] = static_cast<float>(i);
    }

    // Zaman ölçümü
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    
    // Ana döngü - batch işleme
    for (int batch = 0; batch < NUM_BATCHES; batch++) {
        // H2D transferi (batch)
        hipMemcpyAsync(d_input, h_input, batch_size, hipMemcpyHostToDevice, stream);
        
        // Kernel çalıştır (batch)
        int blockSize = 256;
        int gridSize = (batch_size / sizeof(float) + blockSize - 1) / blockSize;
        dummyKernel<<<gridSize, blockSize, 0, stream>>>(d_input, d_output, batch_size / sizeof(float));
        
        // D2H transferi (batch)
        hipMemcpyAsync(h_output, d_output, batch_size, hipMemcpyDeviceToHost, stream);
    }
    
    hipStreamSynchronize(stream);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    
    float totalTime;
    hipEventElapsedTime(&totalTime, start, stop);
    
    float throughput = (6000 * 1000.0f) / totalTime;
    std::cout << "Batch Processing - Toplam Süre: " << totalTime << " ms, Throughput: " << throughput << " img/s\n";
    
    // Temizlik
    hipHostFree(h_input);
    hipHostFree(h_output);
    hipFree(d_input);
    hipFree(d_output);
    hipStreamDestroy(stream);
    hipEventDestroy(start);
    hipEventDestroy(stop);
}

int main()
{
    //runSynchronous();
    //runAsynchronous();
    //runCUDAGraph();
    //runMultiStream();
    //runOptimized();
    //optimizeMemoryTransfers();
    //runOptimizedPipeline();
    //runOptimizedGraph();
    //runOptimizedPipelineFixed();


    std::cout << "=== Multi-Threaded Pipeline Testi ===\n";
    runMultiThreadedPipeline();
    
    std::cout << "\n=== Batch Processing Testi ===\n";
    runBatchProcessing();
    

    return 0;
}